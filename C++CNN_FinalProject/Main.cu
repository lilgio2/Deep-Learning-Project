#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
using namespace std;


int main(int argc, const char* argv[])
{
	ifstream datafile;
	datafile.open("data.csv");
	
	string fileline;
	int count = 0;
	int imagenumber = 0;
	int num;

	vector<int> imagerow;
	vector< vector<int> > image;
	vector< vector< vector<int> > > images;
	while (getline(datafile, fileline))
	{
		for (int i = 0; i < fileline.size(); ++i)
		{
			if (i % 2 == 0)
			{
				num = (int)fileline[i] - 48;
				imagerow.push_back(num);
			}
			else
			{
				continue;
			}
		}
		image.push_back(imagerow);
		imagerow.clear();
		++count;
		if (count == 5)
		{
			images.push_back(image);
			image.clear();
			count = 0;
		}
	}

	for (int i = 0; i < images.size(); ++i)
	{
		for (int j = 0; j < images[i].size(); ++j)
		{
			for (int k = 0; k < images[i][j].size(); ++k)
			{
				cout << images[i][j][k];
			}
			cout << endl;
		}
		cout << "-----" << endl;
	}

	vector<int> filterrow1;
	vector<int> filterrow2;
	vector< vector<int> > filter1;
	vector< vector<int> > filter2;
	vector< vector<int> > filter3;
	vector< vector<int> > filter4;
	vector< vector<int> > filter5;
	vector< vector<int> > filter6;
	vector< vector<int> > filter7;
	vector< vector<int> > filter8;
	vector< vector<int> > filter9;
	vector< vector<int> > filter10;


	//Filter 1 Right Side
	filterrow1.push_back(0);
	filterrow1.push_back(1);

	filterrow2.push_back(1);
	filterrow2.push_back(0);

	filter1.push_back(filterrow1);
	filter1.push_back(filterrow1);

	//Filter 2 Left Side
	filter2.push_back(filterrow2);
	filter2.push_back(filterrow2);

	//Filter 3 Top
	filterrow1[0] = 1;	//1 1
	filterrow2[0] = 0;	//0 0

	filter3.push_back(filterrow1);
	filter3.push_back(filterrow2);

	//Filter 4 Bottom
	filter4.push_back(filterrow2);
	filter4.push_back(filterrow1);

	//Filter 5 BL Open
	//filterrow1		  1 1
	filterrow2[1] = 1;	//0 1

	filter5.push_back(filterrow1);
	filter5.push_back(filterrow2);

	//Filter 6 TL Open
	filter6.push_back(filterrow2);
	filter6.push_back(filterrow1);


	//Filter 7 BR Open
	filterrow2[0] = 1; //1 1
	filterrow2[1] = 0; //1 0

	filter7.push_back(filterrow1);
	filter7.push_back(filterrow2);

	//Filter 8 TR Open
	filter8.push_back(filterrow2);
	filter8.push_back(filterrow1);

	//Filter 9 TL BR Open
	filterrow1[0] = 0;	//0 1
	//filterrow2		  1 0

	filter9.push_back(filterrow1);
	filter9.push_back(filterrow2);

	//Filter 10 TR BL Open
	filter10.push_back(filterrow2);
	filter10.push_back(filterrow1);







}


vector< vector< vector<int> > > convolve()
{
	vector< vector< vector<int> > > convolvedImages;
	return convolvedImages;
}