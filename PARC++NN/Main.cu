
#include <iostream>
#include <chrono>
#include "Neuron.cuh"
#include "Matrix.cuh"
#include "NeuralNetwork.cuh"
#include <fstream>
#include <string>


using namespace std;
using namespace std::chrono;

int main(int argc, char** argv)
{
	//test neurons
	/*
	Neuron* n = new Neuron(0.9);
	cout << "Val: " << n->getVal() << endl;
	cout << "ActivatedVal: " << n->getActivatedVal() << endl;
	cout << "DerivedVal: " << n->getDerivedVal() << endl;
	*/

	//test random matrix weights creation and transposing
	/*
	Matrix* m = new Matrix(3, 2, true);
	m->printToConsole();

	cout << "--------------------------------------------------" << endl;

	Matrix* mT = m->transpose();
	mT->printToConsole();
	*/

	//test network creation with input
	/*
	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(3);

	vector<double> input;
	input.push_back(1.0);
	input.push_back(0.0);
	input.push_back(1.0);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);

	nn->printToConsole();
	*/

	//check feedforward is calculating correctly through each layer
	/*
	vector<double> input;
	input.push_back(1);
	input.push_back(0);
	input.push_back(1);

	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(1);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);
	nn->feedForward();
	nn->printToConsole();
	*/

	//check error calculations
	/*
	vector<double> input;
	input.push_back(1);
	input.push_back(0);
	input.push_back(1);

	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(3);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);
	nn->setCurrentTarget(input);
	nn->feedForward();
	nn->setErrors();

	nn->printToConsole();

	cout << "Total Error: " << nn->getTotalError() << endl;
	*/

	//test backprop
	/*
	vector<double> input;
	input.push_back(1);
	input.push_back(0);
	input.push_back(1);

	vector<int> topology;
	topology.push_back(3);
	topology.push_back(2);
	topology.push_back(3);

	NeuralNetwork* nn = new NeuralNetwork(topology);
	nn->setCurrentInput(input);
	nn->setCurrentTarget(input);

	//training process
	for (int i = 0; i < 100000; ++i)
	{
		cout << "Epoch: " << i + 1 << endl;
		nn->feedForward();
		nn->setErrors();
		cout << "Total Error: " << nn->getTotalError() << endl;
		nn->backPropagation();
	}
	*/

ifstream datafile;
datafile.open("data.csv");

string fileline;
int count = 0;
int imagenumber = 0;
double num;


auto start = high_resolution_clock::now();




vector<double> input;
vector< vector<double> > inputs;
while (getline(datafile, fileline))
{
	for (int i = 0; i < fileline.size(); ++i)
	{
		if (i % 2 == 0)
		{
			num = (double)fileline[i] - 48;
			input.push_back(num);
		}
		else
		{
			continue;
		}
	}
	++count;
	if (count == 5)
	{
		inputs.push_back(input);
		input.clear();
		count = 0;
	}
}

vector<double> target;
vector< vector<double> > targets;
for (int i = 0; i < 10; ++i)
{
	target.push_back(0);
}
for (double i = 0.00; i < 10.00; i++)
{
	if (i != 0)
	{
		target[i - 1] = 0;
		target[i] = 1;
	}
	else
	{
		target[(int)i] = 1;
	}
	for (int j = 0; j < 5; ++j)
	{
		targets.push_back(target);
	}
	cout << i << endl;
}

vector<int> topology;
topology.push_back(25);
topology.push_back(23);
topology.push_back(21);
topology.push_back(19);
topology.push_back(17);
topology.push_back(15);
topology.push_back(13);
topology.push_back(11);
topology.push_back(10);


NeuralNetwork* nn = new NeuralNetwork(topology);
//nn->setCurrentInput(input);
//nn->setCurrentTarget(target);

//training process
for (int i = 0; i < 100; ++i)
{
	for (int j = 0; j < inputs.size(); ++j)
	{
		nn->setCurrentInput(inputs[j]);
		nn->setCurrentTarget(targets[j]);
		nn->feedForward();
		nn->setErrors();
		nn->backPropagation();
		if (i % 10 == 0)
		{
			cout << "----------------------------------------" << endl;
			cout << "OUTPUT " << j + 1 << ": ";
			nn->printOutputToConsole();

			cout << "TARGET " << j + 1 << ": ";
			nn->printTargetToConsole();
			cout << "----------------------------------------" << endl;
			cout << endl;
		}
	}
	//cout << "Epoch: " << i + 1 << endl;
	//nn->feedForward();
	//nn->setErrors();
	//cout << "Total Error: " << nn->getTotalError() << endl;
	//nn->backPropagation();

	//cout << "----------------------------------------" << endl;
	//cout << "OUTPUT: ";
	//nn->printOutputToConsole();

	//cout << "TARGET: ";
	//nn->printTargetToConsole();
	//cout << "----------------------------------------" << endl;
	//cout << endl;

}

//nn->printHistoricalErrors();

	delete nn;

	auto stop = high_resolution_clock::now();

	auto duration = duration_cast<microseconds>(stop - start);

	cout << "CUDA" << endl;
	cout << "Time taken by function: "
		<< duration.count() / 1000000 << " seconds" << endl;

	return 0;
}